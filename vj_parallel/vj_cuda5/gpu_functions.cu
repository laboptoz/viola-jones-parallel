#include "hip/hip_runtime.h"
#include "gpu_functions.cuh"
#include <stdio.h>

__global__  void evalWeakClassifier(int variance_norm_factor, int p_offset, int tree_index, int w_index, int r_index, int* stage_sum_array_cu, int* stages_array_cu,
									int* tree_thresh_array_cu, int* scaled_rectangles_array_cu, int* weights_array_cu, int* alpha1_array_cu,
                                    int* alpha2_array_cu, int* sum_data_cu){
	for(int j = 0; j < stages_array_cu[threadIdx.x]; j++ ){
		int t = tree_thresh_array_cu[tree_index] * variance_norm_factor;

		int sum = *(sum_data_cu + scaled_rectangles_array_cu[r_index] + p_offset)
				 - *(sum_data_cu + scaled_rectangles_array_cu[r_index + 1] + p_offset)
		 		 - *(sum_data_cu + scaled_rectangles_array_cu[r_index + 2] + p_offset)
		 		 + *(sum_data_cu + scaled_rectangles_array_cu[r_index + 3] + p_offset)
    			 * weights_array_cu[w_index];

    	sum += (*(sum_data_cu + scaled_rectangles_array_cu[r_index+4] + p_offset)
            - *(sum_data_cu + scaled_rectangles_array_cu[r_index+5] + p_offset)
            - *(sum_data_cu + scaled_rectangles_array_cu[r_index + 6] + p_offset)
            + *(sum_data_cu + scaled_rectangles_array_cu[r_index + 7] + p_offset))
        	* weights_array_cu[w_index + 1];

    	if ((scaled_rectangles_array_cu[r_index+8] != 0))
        	sum += (*(sum_data_cu + scaled_rectangles_array_cu[r_index+8] + p_offset)
                - *(sum_data_cu + scaled_rectangles_array_cu[r_index + 9] + p_offset)
                - *(sum_data_cu + scaled_rectangles_array_cu[r_index + 10] + p_offset)
                + *(sum_data_cu + scaled_rectangles_array_cu[r_index + 11] + p_offset))
            	* weights_array_cu[w_index + 2];

    	if(sum >= t)
			stage_sum_array_cu[threadIdx.x]+=alpha2_array_cu[tree_index];
 		else
			stage_sum_array_cu[threadIdx.x]+= alpha1_array_cu[tree_index];
	}

	tree_index++;
	w_index+=3;
	r_index+=12;

}

